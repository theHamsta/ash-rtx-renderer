
#include <hip/hip_runtime.h>

extern "C" {

__global__ void simple(int width, int height, float time,
                       hipSurfaceObject_t tex) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  if (x >= width || y >= height) {
    return;
  }
  unsigned char val = (x % 255) * time;
  surf2Dwrite(uchar4{val, 0, 0, 255}, tex, x * 4, y);
}

}
