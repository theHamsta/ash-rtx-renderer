
#include <hip/hip_runtime.h>

extern "C" {

__global__ void simple(int width, int height, float time,
                       hipSurfaceObject_t tex) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  if (x >= width || y >= height) {
    return;
  }
  unsigned char val = x % 255;
  surf2Dwrite(uchar4{val, 0, 0, 255}, tex, x, y);
}

}
