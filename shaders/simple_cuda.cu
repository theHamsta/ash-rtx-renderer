
#include <hip/hip_runtime.h>

extern "C" {

__global__ void simple(int width, int height, float time,
                       hipSurfaceObject_t tex) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  if (x >= width || y >= height) {
    return;
  }
  surf2Dwrite(uchar4{255, 100, 100, 255}, tex, x, y);
}
}
